#include <stdio.h>
#include <hip/hip_runtime.h>

// kernel which send instruction to gpu to parallely process it
__global__ void dkernel() {
	printf("Hello World.\n");
}

int main() {
	dkernel<<<1,1>>>();
//	Synchronizes CPU and GPU i.e. CPU doesn't continue with further code until GPU is done with processing all instructions.
//	cudaDeviceSynchronize();
	return 0;
}
