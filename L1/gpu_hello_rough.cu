#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel1(){
	printf("Hello World 1.\n");
}

__global__ void dkernel2(){
	printf("Hello World 2.\n");
}

__global__ void dkernel3(){
	printf("Hello World 3.\n");
}
int main(){
	dkernel1<<<1, 1>>>();
	
	dkernel2<<<1, 1>>>();
	hipDeviceSynchronize();
	
	dkernel3<<<1, 1>>>();
	hipDeviceSynchronize();
	return 0;
}
