#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100
__global__ void func(){
	printf("%d\n", threadIdx.x * threadIdx.x);
}

int main(){
	func<<<1, N>>>();
	hipDeviceSynchronize();
	return 0;
}
