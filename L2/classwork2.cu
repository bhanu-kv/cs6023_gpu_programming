#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 100

__global__ void func(int* a){
	a[threadIdx.x] = threadIdx.x * threadIdx.x;
}

int main(){
	int a[N], *da;
	
	hipMalloc(&da, N*sizeof(int));
	func<<<1, N>>>(da);
	//cudaDeviceSynchronize(); // This doesn't matter since cudaMemcpy Synchronizes
	
	hipMemcpy(a, da, N*sizeof(int), hipMemcpyDeviceToHost);
	
	for(int i = 0; i<N; i++){
		printf("%d\n", a[i]);
	}
	return 0;
}
