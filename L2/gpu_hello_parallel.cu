#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(){
	printf("Hello World.\n");
}

int main(){
//	prints 32 different Hello World which comes from 32 different threads
//	1 is the number of thread blocks that are being assigned
//	We can maximum assign 1024 threads according to the GPU limit on my PC
//	Number of thread blocks can be unlimited
//	If more number of thread blocks are assigned than available then the cores will be processed sequentially from built-in algorithm
	dkernel<<<1, 32>>>();
	hipDeviceSynchronize();
	return 0;
}
