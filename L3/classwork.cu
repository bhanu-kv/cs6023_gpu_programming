#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel1(int *gpuarr, int N){
	unsigned id = threadIdx.x;
	
	if (id < N){
		gpuarr[id] = 0;
	}
}

__global__ void dkernel2(int *gpuarr, int N){
	unsigned id = threadIdx.x;
	
	if (id < N){
		gpuarr[id] += id;
	}
}

int main(){
	int N = 8000;
	int a[N], *gpuarr;
	
	hipMalloc(&gpuarr, sizeof(int) * N);
	dkernel1<<<1, N>>>(gpuarr, N);
	hipDeviceSynchronize();
	
	dkernel2<<<1, N>>>(gpuarr, N);
	
	hipMemcpy(a, gpuarr, sizeof(int) * N, hipMemcpyDeviceToHost);
	
	for(int i = 0; i < N; i++){
		printf("%d\n", a[i]);
	}
	
	return 0;
}
