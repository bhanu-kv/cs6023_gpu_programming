#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(char *arr, int arrlen){
	unsigned id = threadIdx.x;
	
	// make sure only correct threads i.e. correct memory location is updated
	if (id < arrlen){
		++arr[id];
	}
}

int main(){
	// Assigning a pointer to gpu array
	char cpuarr[] = "Gdkkn\x1fVnqkc-", *gpuarr;
	
	// Assigns memory to gpu array and copy cpu string to gpu
	hipMalloc(&gpuarr, sizeof(char) * (1 + strlen(cpuarr)));
	hipMemcpy(gpuarr, cpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyHostToDevice);
	
	// Run kernel to change gpu string
	dkernel<<<1, 32>>>(gpuarr, strlen(cpuarr));
	
	// Synchronization not needed before memcpy
	hipDeviceSynchronize();
	
	// Copy gpu array to cpu and print
	hipMemcpy(cpuarr, gpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyDeviceToHost);
	printf("%s", cpuarr);
	
	return 0;
}
