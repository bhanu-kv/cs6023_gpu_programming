#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>

__global__ void dkernel_square(int *arr, int N){
	unsigned id = threadIdx.x;
	
	if (id < N){
		arr[id] = id*id;
	}
}

__global__ void dkernel_cube(int *arr, int N){
	unsigned id = threadIdx.x;
	
	if (id < N){
		arr[id] = id*id*id;
	}
}

int main(){
	ifstream read_file("homework_numbers.txt");
	assert(read_file.is_open());
	
	int* arr_x, gpuarr_x;
	
	int N = 0;
	while(!read_file.eof())
	{
		read_file >> arr_x[i];
		N++;
	}
	
	read_file.close();
	
	hipMalloc(&gpuarr_x, sizeof(int) * N);
	hipMemcpy(gpuarr_x, arr_x, sizeof(int) * N, hipMemcpyHostToDevice);
	dkernel<<<1, N>>>(gpuarr_x, N);
	
	hipMemcpy(arr_x, gpuarr_x, sizeof(int) * N, hipMemcpyDeviceToHost);
	
	ifstream read_file("homework_numbers.txt");
	assert(read_file.is_open());
	
	int* arr_y, gpuarr_y;
	
	int M = 0;
	while(!read_file.eof())
	{
		read_file >> arr_y[i];
		M++;
	}
	
	read_file.close();
	
	hipMalloc(&gpuarr_y, sizeof(int) * M);
	hipMemcpy(gpuarr_y, arr_y, sizeof(int) * M, hipMemcpyHostToDevice);
	dkernel<<<1, N>>>(gpuarr_y, M);
	
	hipMemcpy(arr_y, gpuarr_y, sizeof(int) * M, hipMemcpyDeviceToHost);
	
	return 0;
}
